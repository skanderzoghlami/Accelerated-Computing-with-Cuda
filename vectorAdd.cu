
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x ; 
  int stride = blockDim.x * gridDim.x ; 
  for(int i = idx; i < N; i+=stride )
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  size_t number_of_threads = 256 ;
  size_t number_of_blocks  = (N + number_of_threads -1 ) / number_of_threads;

  hipError_t syncErr , asyncErr ;


  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a,size);
  hipMallocManaged(&b,size);
  hipMallocManaged(&c,size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  addVectorsInto<<<number_of_blocks,number_of_threads>>>(c, a, b, N);
  syncErr = hipDeviceSynchronize();
  asyncErr = hipGetLastError();
  checkElementsAre(7, c, N);
  if (syncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(syncErr));
  if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));
  hipFree(a);
  hipFree(b);
  hipFree(c);
}

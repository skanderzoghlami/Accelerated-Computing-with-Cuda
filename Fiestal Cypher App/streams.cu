#include "hip/hip_runtime.h"
#include <cstdint>
#include <iostream>
#include "helpers.cuh"
#include "encryption.cuh"

void encrypt_cpu(uint64_t * data, uint64_t num_entries, 
                 uint64_t num_iters, bool parallel=true) {

    #pragma omp parallel for if (parallel)
    for (uint64_t entry = 0; entry < num_entries; entry++)
        data[entry] = permute64(entry, num_iters);
}

__global__ 
void decrypt_gpu(uint64_t * data, uint64_t num_entries, 
                 uint64_t num_iters) {

    const uint64_t thrdID = blockIdx.x*blockDim.x+threadIdx.x;
    const uint64_t stride = blockDim.x*gridDim.x;

    for (uint64_t entry = thrdID; entry < num_entries; entry += stride)
        data[entry] = unpermute64(data[entry], num_iters);
}

bool check_result_cpu(uint64_t * data, uint64_t num_entries,
                      bool parallel=true) {

    uint64_t counter = 0;

    #pragma omp parallel for reduction(+: counter) if (parallel)
    for (uint64_t entry = 0; entry < num_entries; entry++)
        counter += data[entry] == entry;

    return counter == num_entries;
}

int main (int argc, char * argv[]) {

    const char * encrypted_file = "/dli/task/encrypted";

    Timer timer;

    const uint64_t num_entries = 1UL << 26;
    const uint64_t num_iters = 1UL << 10;
    const bool openmp = true;

    uint64_t * data_cpu, * data_gpu;
    hipHostMalloc(&data_cpu, sizeof(uint64_t)*num_entries);
    hipMalloc    (&data_gpu, sizeof(uint64_t)*num_entries);
    
    const uint64_t num_streams = 1000;
    // Array of Streams
    hipStream_t streams[num_streams];
    
    for(uint64_t i = 0 ; i < num_streams ; ++i)
        hipStreamCreate(&streams[i]);
    // Chunk Size 
    uint64_t chunk_size = sdiv(num_entries , num_streams); 
    
    check_last_error();

    if (!encrypted_file_exists(encrypted_file)) {
        encrypt_cpu(data_cpu, num_entries, num_iters, openmp);
        write_encrypted_to_file(encrypted_file, data_cpu, sizeof(uint64_t)*num_entries);
    } else {
        read_encrypted_from_file(encrypted_file, data_cpu, sizeof(uint64_t)*num_entries);
    }

    timer.start();
    
    
    // Copy Compute OverLap Starts Here
    for(uint64_t stream = 0 ; stream < num_streams ; ++stream){
    
    const uint64_t lower = stream * chunk_size;
    const uint64_t upper = min(lower + chunk_size  , num_entries);
    const uint64_t width = upper - lower;
    
    hipMemcpyAsync(data_gpu+lower, data_cpu+lower, 
               sizeof(uint64_t)*width, hipMemcpyHostToDevice, streams[stream]);
    check_last_error();

    decrypt_gpu<<<80*32, 64,0,streams[stream]>>>(data_gpu+lower, width, num_iters);
    check_last_error();

    hipMemcpyAsync(data_cpu+lower, data_gpu+lower, 
               sizeof(uint64_t)*width, hipMemcpyDeviceToHost , streams[stream]);
    
    }
    timer.stop("total time on GPU");
    check_last_error();
    
    hipDeviceSynchronize();
    const bool success = check_result_cpu(data_cpu, num_entries, openmp);
    
   
    std::cout << "STATUS: test " 
              << ( success ? "passed" : "failed")
              << std::endl;

    hipHostFree(data_cpu);
    hipFree    (data_gpu);
    check_last_error();
}
